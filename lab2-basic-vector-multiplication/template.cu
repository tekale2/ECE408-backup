#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                          \
  do {                                                                         \
    hipError_t err = stmt;                                                    \
    if (err != hipSuccess) {                                                  \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                              \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));           \
      return -1;                                                               \
    }                                                                          \
  } while (0)

// Compute C = A * B
__global__ void matrixMultiply(float *A, float *B, float *C, int numARows,
                               int numAColumns, int numBRows, int numBColumns,
                               int numCRows, int numCColumns) {
	
  //@@ Insert code to implement matrix multiplication here
int Row = blockIdx.y*blockDim.y + threadIdx.y;
int Column = blockIdx.x*blockDim.x + threadIdx.x;
//checking if matrices are multipliable
	if (numAColumns != numBRows) 
	return;
//Checking the boundry conditions
 if ((Row < numARows) && (Column < numBColumns)) {
	float P = 0.00;
	//comuting and adding elements of C
	for (int k = 0; k < numAColumns; ++k)
    P += A[Row*numAColumns+k] * B[k*numBRows+Column];
	 
   	C[Row*numCColumns+Column] = P;
	
  }


		
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA;
  float *deviceB;
  float *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set this)
  int numThreads; //num threads
  args = wbArg_read(argc, argv);
  //initializing it to 32
	numThreads = 32;
  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA =( float * )wbImport(wbArg_getInputFile(args, 0), &numARows, &numAColumns);
  hostB =( float * )wbImport(wbArg_getInputFile(args, 1), &numBRows, &numBColumns);
  //@@ Set numCRows and numCColumns
  numCRows = numARows;
  numCColumns = numBColumns;
	
  //@@ Allocate the hostC matrix
	
	hostC = (float*) malloc( sizeof(float)*numCRows*numCColumns);
	
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

  wbTime_start(GPU, "Allocating GPU memory.");
	
  //@@ Allocate GPU memory here
	
	hipMalloc((void**)&deviceA, numARows*numAColumns*sizeof(float));
	hipMalloc((void**)&deviceB, numBRows*numBColumns*sizeof(float));
	hipMalloc((void**)&deviceC, numCRows*numCColumns*sizeof(float));
	
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
	
  //@@ Copy memory to the GPU here
	
	hipMemcpy(deviceA, hostA, numARows*numAColumns*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(deviceB, hostB, numBRows*numBColumns*sizeof(float),hipMemcpyHostToDevice);
	
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
	dim3 DimGrid((numCColumns - 1) / numThreads + 1, (numCRows - 1) / numThreads + 1, 1);
    dim3 DimBlock(numThreads , numThreads, 1);
	
	wbTime_start(Compute, "Performing CUDA computation");
	
  //@@ Launch the GPU Kernel here
	
 matrixMultiply<<<DimGrid , DimBlock>>>(deviceA , deviceB , deviceC , numARows , numAColumns, numBRows ,numBColumns , numCRows , numCColumns);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
	hipMemcpy(hostC, deviceC, numCRows*numCColumns*sizeof(float), hipMemcpyDeviceToHost);

  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
	
  //@@ Free the GPU memory here
	hipFree(deviceA);
	hipFree(deviceB);
	hipFree(deviceC);
	
  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
