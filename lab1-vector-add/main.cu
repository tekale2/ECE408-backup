/******************************************************************************
 *cr
 *cr         (C) Copyright 2010-2013 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#include <stdio.h>
#include "support.h"
#include "kernel.cu"

int main(int argc, char**argv) {

    Timer timer;
    hipError_t cuda_ret;
    int n;
	const unsigned int THREADS_PER_BLOCK = 512;
	unsigned int numBlocks;

    // Initialize host variables ----------------------------------------------
    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    if(argc == 1) {
        n = 10000;
    } else if(argc == 2) {
        n = atoi(argv[1]);
    } else {
        printf("\n    Invalid input parameters!"
           "\n    Usage: ./vecadd               # Vector of size 10,000 is used"
           "\n    Usage: ./vecadd <m>           # Vector of size m is used"
           "\n");
        exit(0);
    }

    float* A_h = (float*) malloc( sizeof(float)*n );
    for (unsigned int i=0; i < n; i++) { A_h[i] = (rand()%100)/100.00; }

    float* B_h = (float*) malloc( sizeof(float)*n );
    for (unsigned int i=0; i < n; i++) { B_h[i] = (rand()%100)/100.00; }

    float* C_h = (float*) malloc( sizeof(float)*n );

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    Vector size = %u\n", n);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);
	float* A_d;
	float* B_d;
	float* C_d;
	//Allocating memory on the GPU
	hipMalloc((void**)&A_d, n*sizeof(float));
	hipMalloc((void**)&B_d, n*sizeof(float));
	hipMalloc((void**)&C_d, n*sizeof(float));


    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------

    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    //copying all three vectors from cpu to device
	hipMemcpy(A_d, A_h, sizeof(float)*n, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, sizeof(float)*n, hipMemcpyHostToDevice);

   //wait for all copying to finish before launching the kernel
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
	
	//calculate numBlocks 
	numBlocks = (n - 1)/THREADS_PER_BLOCK + 1;
    // Launch kernel ----------------------------------------------------------

    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);
	
	//Setting up dimensions and launching the kernel
	dim3 gridDim(numBlocks, 1, 1);
	dim3 blockDim(THREADS_PER_BLOCK, 1, 1);
    vecAddKernel<<< gridDim, blockDim >>> (A_d, B_d, C_d, n);

	//wait for kernel stream to finish
    cuda_ret = hipDeviceSynchronize();
	if(cuda_ret != hipSuccess) FATAL("Unable to launch kernel");
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy device variables from host ----------------------------------------

    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);

    hipMemcpy(C_h, C_d, sizeof(float)*n, hipMemcpyDeviceToHost);

	//wait for all copying to finish
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Verify correctness -----------------------------------------------------

    printf("Verifying results..."); fflush(stdout);

    verify(A_h, B_h, C_h, n);

    // Free memory ------------------------------------------------------------

    free(A_h);
    free(B_h);
    free(C_h);

    //Freeing memory on device
	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);

     //yay! successful!

    return 0;
}

